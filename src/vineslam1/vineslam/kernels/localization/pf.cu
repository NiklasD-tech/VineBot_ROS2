#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <vineslam/interface/localization_mapping_interface.hpp>
#include <vineslam/localization/pf.hpp>
#include <vineslam/feature/three_dimensional.hpp>

#include "../gpu_timer.h"

namespace vineslam
{
__device__ void alignPoint(const float& in_x, const float& in_y, const float& cx, const float& cy, const float& angle, float& out_x, float& out_y)
{
  float x1 = in_x - cx;
  float y1 = in_y - cy;

  out_x = x1 * cos(angle) - y1 * sin(angle);
  out_y = x1 * sin(angle) + y1 * cos(angle);

  out_x += cx;
  out_y += cy;
}

__device__ int to1D(const float& x, const float& y, const float& z, const float& origin_x, const float& origin_y, const float& origin_z,
                    const float& resolution, const float& resolution_z, const float& width, const float& length)
{
  int ii = static_cast<int>(std::round(x / resolution + 0.49));
  int jj = static_cast<int>(std::round(y / resolution + 0.49));
  int xx = ii - static_cast<int>(std::round(origin_x / resolution + 0.49));
  int yy = jj - static_cast<int>(std::round(origin_y / resolution + 0.49));
  int zz = static_cast<int>(std::round((z - origin_z) / resolution_z));
  return (xx + (static_cast<int>(std::round(width / resolution + 0.49)) * (yy + (zz * static_cast<int>(std::round(length / resolution + 0.49))))));
}

__device__ uint8_t getBit(const int& pos, const uint8_t& byte)
{
  return ((byte >> pos) & 1U);
}

// Kernel function for corner-based weight calculation
__global__ void cornerWeightCalculation(uint8_t** map, uint32_t* sizes, float* infos, float* ts, float* features, LookUpTable look_up_table, int particles_size,
                                        int corners_size, float* weights)
{
  int particle_idx = threadIdx.x + blockIdx.x * blockDim.x;
  int feature_idx = threadIdx.y + blockIdx.y * blockDim.y;

  if (particle_idx < particles_size)
  {
    if (feature_idx < corners_size)
    {
      // Transform feature to particles' reference frame
      float ft_x = features[feature_idx * 3 + 0] * ts[particle_idx * 12 + 0] + features[feature_idx * 3 + 1] * ts[particle_idx * 12 + 1] +
                   features[feature_idx * 3 + 2] * ts[particle_idx * 12 + 2] + ts[particle_idx * 12 + 9];
      float ft_y = features[feature_idx * 3 + 0] * ts[particle_idx * 12 + 3] + features[feature_idx * 3 + 1] * ts[particle_idx * 12 + 4] +
                   features[feature_idx * 3 + 2] * ts[particle_idx * 12 + 5] + ts[particle_idx * 12 + 10];
      float ft_z = features[feature_idx * 3 + 0] * ts[particle_idx * 12 + 6] + features[feature_idx * 3 + 1] * ts[particle_idx * 12 + 7] +
                   features[feature_idx * 3 + 2] * ts[particle_idx * 12 + 8] + ts[particle_idx * 12 + 11];

      // Compute the topological node where the feature lies
      int i_look_up_idx = (ft_x - look_up_table.look_up_table_min_x_) / look_up_table.look_up_table_res_;
      int j_look_up_idx = (ft_y - look_up_table.look_up_table_min_y_) / look_up_table.look_up_table_res_;
      int index = i_look_up_idx + j_look_up_idx * look_up_table.look_up_table_cols_;

      if (index < look_up_table.look_up_table_size_ && index > 0)
      {
        int node_number = look_up_table.look_up_table_[index];

        if (sizes[node_number] > 0)
        {
          // Align point to match the local grid map reference frame
          float aligned_x, aligned_y;
          alignPoint(ft_x, ft_y, infos[node_number * 10 + 0], infos[node_number * 10 + 1], infos[node_number * 10 + 2], aligned_x, aligned_y);

          // Get the index to access the sub map stored in the topological node
          int local_map_idx = to1D(aligned_x, aligned_y, ft_z, infos[node_number * 10 + 3], infos[node_number * 10 + 4], infos[node_number * 10 + 5],
                                   infos[node_number * 10 + 6], infos[node_number * 10 + 7], infos[node_number * 10 + 8], infos[node_number * 10 + 9]);

          // Get the byte index where the feature boolean is stored
          int byte_number = static_cast<int>(local_map_idx / 8);

          // Get the occupancy result from the obtained index
          if (byte_number < sizes[node_number] && byte_number >= 0)
          {
            int bit_number = local_map_idx % 8;
            float val = (float)getBit(bit_number, map[node_number][byte_number]);
            atomicAdd(&weights[particle_idx], val);
          }
        }
      }
    }
  }
}

// Kernel function for planar-based weight calculation
__global__ void planarWeightCalculation(uint8_t** map, uint32_t* sizes, float* infos, float* ts, float* features, LookUpTable look_up_table, int particles_size,
                                        int planars_size, float* weights)
{
  int particle_idx = threadIdx.x + blockIdx.x * blockDim.x;
  int feature_idx = threadIdx.y + blockIdx.y * blockDim.y;

  if (particle_idx < particles_size)
  {
    if (feature_idx < planars_size)
    {
      // Transform feature to particles' reference frame
      float ft_x = features[feature_idx * 3 + 0] * ts[particle_idx * 12 + 0] + features[feature_idx * 3 + 1] * ts[particle_idx * 12 + 1] +
                   features[feature_idx * 3 + 2] * ts[particle_idx * 12 + 2] + ts[particle_idx * 12 + 9];
      float ft_y = features[feature_idx * 3 + 0] * ts[particle_idx * 12 + 3] + features[feature_idx * 3 + 1] * ts[particle_idx * 12 + 4] +
                   features[feature_idx * 3 + 2] * ts[particle_idx * 12 + 5] + ts[particle_idx * 12 + 10];
      float ft_z = features[feature_idx * 3 + 0] * ts[particle_idx * 12 + 6] + features[feature_idx * 3 + 1] * ts[particle_idx * 12 + 7] +
                   features[feature_idx * 3 + 2] * ts[particle_idx * 12 + 8] + ts[particle_idx * 12 + 11];

      // Compute the topological node where the feature lies
      int i_look_up_idx = (ft_x - look_up_table.look_up_table_min_x_) / look_up_table.look_up_table_res_;
      int j_look_up_idx = (ft_y - look_up_table.look_up_table_min_y_) / look_up_table.look_up_table_res_;
      int index = i_look_up_idx + j_look_up_idx * look_up_table.look_up_table_cols_;

      if (index < look_up_table.look_up_table_size_ && index > 0)
      {
        int node_number = look_up_table.look_up_table_[index];

        if (sizes[node_number] > 0)
        {
          // Align point to match the local grid map reference frame
          float aligned_x, aligned_y;
          alignPoint(ft_x, ft_y, infos[node_number * 10 + 0], infos[node_number * 10 + 1], infos[node_number * 10 + 2], aligned_x, aligned_y);

          // Get the index to access the sub map stored in the topological node
          int local_map_idx = to1D(aligned_x, aligned_y, ft_z, infos[node_number * 10 + 3], infos[node_number * 10 + 4], infos[node_number * 10 + 5],
                                   infos[node_number * 10 + 6], infos[node_number * 10 + 7], infos[node_number * 10 + 8], infos[node_number * 10 + 9]);

          // Get the byte index where the feature boolean is stored
          int byte_number = static_cast<int>(sizes[node_number] / 2) + static_cast<int>(local_map_idx / 8);

          // Get the occupancy result from the obtained index
          if (byte_number >= static_cast<int>(sizes[node_number] / 2) && byte_number < sizes[node_number])
          {
            int bit_number = local_map_idx % 8;
            float val = (float)getBit(bit_number, map[node_number][byte_number]);
            atomicAdd(&weights[particle_idx], val);
          }
        }
      }
    }
  }
}

__global__ void weightRefinement(float* corner_weights, float* planar_weights, int particles_size, float sigma_corner_matching, float sigma_planar_matching)
{
  int particle_idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (particle_idx < particles_size)
  {
    float normalizer_corner = 1.0 / (sigma_corner_matching * sqrt(M_2PI));
    float w_corners = corner_weights[particle_idx];
    corner_weights[particle_idx] = (normalizer_corner * (exp((1.0 / sigma_corner_matching) * w_corners)));

    float normalizer_planar = 1.0 / (sigma_planar_matching * sqrt(M_2PI));
    float w_planars = planar_weights[particle_idx];
    planar_weights[particle_idx] = (normalizer_planar * (exp((1.0 / sigma_planar_matching) * w_planars)));
  }
}

void pfUpdate(const std::vector<Corner>& corners, const std::vector<Planar>& planars, std::vector<Particle>& particles,
              LocalizationMappingInterface* localization_mapping_interface)
{
  // Declare timer
  GpuTimer timer;

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // Kernel settings
  int block_dim_per_component = 16;
  int particles_size = particles.size();
  int corners_size = corners.size();
  int planars_size = planars.size();
  float simple_weight_factor = 20.0;
  float sigma_corner_matching = static_cast<float>(corners_size) / simple_weight_factor;
  float sigma_planar_matching = static_cast<float>(planars_size) / simple_weight_factor;

  // Allocate memory on the device
  // uint8_t** d_array: localization and mapping interface structure that store the topological map into a matrix
  // uint32_t* d_sizes: array that stores the size of each topological node (allows knowing if they are allocated)
  // float*    d_infos: information required for the localization process
  //                    organized as a 1D array with 10 data fields for each node
  //                    [center_x, center_y, angle, origin_x, origin_y, origin_z, res, res_z, width, length]
  // float* d_transformations: array that stores a homogeneous transformation for each particle (12 fields per particle)
  // float* d_corners:         array that stores the corners features [x1, y1, z1, x2, y2, z2, ..., xn, yn, zn]
  // float* d_planars:         array that stores the planars features [x1, y1, z1, x2, y2, z2, ..., xn, yn, zn]
  // LookUpTable* d_look_up_table: structure that holds the necessary data to index the topological nodes from 3D
  //                               coordinates
  // float*       d_weights: output particle weights (1-to-1)

  // ****************************
  // *** Input Map
  // ****************************
  timer.Start();
  uint8_t** tmp = (uint8_t**)malloc(localization_mapping_interface->number_of_nodes_ * sizeof(uint8_t*));
  uint8_t** d_array = NULL;
  for (int i = 0; i < localization_mapping_interface->number_of_nodes_; i++)
  {
    if (localization_mapping_interface->sizes_[i] > 0)
    {
      hipMalloc((void**)&tmp[i], localization_mapping_interface->sizes_[i] * sizeof(uint8_t));
    }
  }
  hipMalloc((void**)&d_array, localization_mapping_interface->number_of_nodes_ * sizeof(uint8_t*));
  for (int i = 0; i < localization_mapping_interface->number_of_nodes_; i++)
  {
    if (localization_mapping_interface->sizes_[i] > 0)
    {
      hipMemcpy(tmp[i], localization_mapping_interface->array_[i], localization_mapping_interface->sizes_[i] * sizeof(uint8_t), hipMemcpyHostToDevice);
    }
  }
  hipMemcpy(d_array, tmp, localization_mapping_interface->number_of_nodes_ * sizeof(uint8_t*), hipMemcpyHostToDevice);

  // ****************************
  // *** Input Sizes
  // ****************************
  uint32_t* d_sizes = NULL;
  err = hipMalloc((void**)&d_sizes, localization_mapping_interface->number_of_nodes_ * sizeof(uint32_t));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector d_sizes (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err =
      hipMemcpy(d_sizes, localization_mapping_interface->sizes_, localization_mapping_interface->number_of_nodes_ * sizeof(uint32_t), hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector sizes from host to device (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // ****************************
  // *** Input Infos
  // ****************************
  float* d_info = NULL;
  err = hipMalloc((void**)&d_info, localization_mapping_interface->number_of_nodes_ * 10 * sizeof(float));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector d_info (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err =
      hipMemcpy(d_info, localization_mapping_interface->info_, localization_mapping_interface->number_of_nodes_ * 10 * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector info from host to device (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // ****************************
  // *** Input Transformations
  // ****************************
  float* h_transformations = (float*)malloc(particles_size * 12 * sizeof(float));
  for (int i = 0; i < particles_size; i++)
  {
    h_transformations[i * 12 + 0] = particles[i].tf_.R_array_[0];
    h_transformations[i * 12 + 1] = particles[i].tf_.R_array_[1];
    h_transformations[i * 12 + 2] = particles[i].tf_.R_array_[2];
    h_transformations[i * 12 + 3] = particles[i].tf_.R_array_[3];
    h_transformations[i * 12 + 4] = particles[i].tf_.R_array_[4];
    h_transformations[i * 12 + 5] = particles[i].tf_.R_array_[5];
    h_transformations[i * 12 + 6] = particles[i].tf_.R_array_[6];
    h_transformations[i * 12 + 7] = particles[i].tf_.R_array_[7];
    h_transformations[i * 12 + 8] = particles[i].tf_.R_array_[8];
    h_transformations[i * 12 + 9] = particles[i].tf_.t_array_[0];
    h_transformations[i * 12 + 10] = particles[i].tf_.t_array_[1];
    h_transformations[i * 12 + 11] = particles[i].tf_.t_array_[2];
  }
  float* d_transformations = NULL;
  err = hipMalloc((void**)&d_transformations, particles_size * 12 * sizeof(float));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector d_transformations (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_transformations, h_transformations, particles_size * 12 * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr,
            "Failed to copy vector transformations from host to device (localization and mapping interface) (error "
            "code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // ****************************
  // *** Input Features
  // ****************************
  // Corners
  float* h_corners = (float*)malloc(corners_size * 3 * sizeof(float));
  for (int i = 0; i < corners_size; i++)
  {
    h_corners[i * 3 + 0] = corners[i].pos_.x_;
    h_corners[i * 3 + 1] = corners[i].pos_.y_;
    h_corners[i * 3 + 2] = corners[i].pos_.z_;
  }
  float* d_corners = NULL;
  err = hipMalloc((void**)&d_corners, corners_size * 3 * sizeof(float));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector d_corners (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_corners, h_corners, corners_size * 3 * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr,
            "Failed to copy vector corners from host to device (localization and mapping interface) (error "
            "code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  // Planars
  float* h_planars = (float*)malloc(planars_size * 3 * sizeof(float));
  for (int i = 0; i < planars_size; i++)
  {
    h_planars[i * 3 + 0] = planars[i].pos_.x_;
    h_planars[i * 3 + 1] = planars[i].pos_.y_;
    h_planars[i * 3 + 2] = planars[i].pos_.z_;
  }
  float* d_planars = NULL;
  err = hipMalloc((void**)&d_planars, planars_size * 3 * sizeof(float));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector d_planars (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_planars, h_planars, planars_size * 3 * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr,
            "Failed to copy vector planars from host to device (localization and mapping interface) (error "
            "code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // ****************************
  // *** Lookup table
  // ****************************
  LookUpTable d_look_up_table;
  err = hipMalloc((void**)&d_look_up_table.look_up_table_, localization_mapping_interface->look_up_table_data_.look_up_table_size_ * sizeof(uint32_t));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector d_look_up_table (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_look_up_table.look_up_table_, localization_mapping_interface->look_up_table_data_.look_up_table_,
                   localization_mapping_interface->look_up_table_data_.look_up_table_size_ * sizeof(uint32_t), hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr,
            "Failed to copy vector lookup_table from host to device (localization and mapping interface) (error "
            "code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  d_look_up_table.look_up_table_size_ = localization_mapping_interface->look_up_table_data_.look_up_table_size_;
  d_look_up_table.look_up_table_min_x_ = localization_mapping_interface->look_up_table_data_.look_up_table_min_x_;
  d_look_up_table.look_up_table_min_y_ = localization_mapping_interface->look_up_table_data_.look_up_table_min_y_;
  d_look_up_table.look_up_table_res_ = localization_mapping_interface->look_up_table_data_.look_up_table_res_;
  d_look_up_table.look_up_table_cols_ = localization_mapping_interface->look_up_table_data_.look_up_table_cols_;

  // ****************************
  // *** Output weights
  // ****************************
  float* h_corner_weights = (float*)malloc(particles_size * sizeof(float));
  float* h_planar_weights = (float*)malloc(particles_size * sizeof(float));
  for (int i = 0; i < particles_size; i++)
  {
    h_corner_weights[i] = 0.0;
    h_planar_weights[i] = 0.0;
  }
  // Corners
  float* d_corner_weights = NULL;
  err = hipMalloc((void**)&d_corner_weights, particles_size * sizeof(float));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector d_corner_weights (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_corner_weights, h_corner_weights, particles_size * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr,
            "Failed to copy vector corner weights from host to device (localization and mapping interface) (error "
            "code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  // Planars
  float* d_planar_weights = NULL;
  err = hipMalloc((void**)&d_planar_weights, particles_size * sizeof(float));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector d_planar_weights (localization and mapping interface) (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_planar_weights, h_planar_weights, particles_size * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr,
            "Failed to copy vector planar weights from host to device (localization and mapping interface) (error "
            "code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  timer.Stop();

  // Call the kernel
  // ----------------------------
  // --- Kernel call (!!!)
  // ----------------------------
  timer.Start();
  dim3 block_dim(block_dim_per_component, block_dim_per_component);

  dim3 grid_dim_for_corners((particles_size + block_dim.x - 1) / block_dim.x, (corners_size + block_dim.y - 1) / block_dim.y);
  cornerWeightCalculation<<<grid_dim_for_corners, block_dim>>>(d_array, d_sizes, d_info, d_transformations, d_corners,
                                                               d_look_up_table, particles_size, corners_size,
                                                               d_corner_weights);

  dim3 grid_dim_for_planars((particles_size + block_dim.x - 1) / block_dim.x, (planars_size + block_dim.y - 1) / block_dim.y);
  planarWeightCalculation<<<grid_dim_for_planars, block_dim>>>(d_array, d_sizes, d_info, d_transformations, d_planars, d_look_up_table, particles_size,
                                                               planars_size, d_planar_weights);

  weightRefinement<<<(particles_size + block_dim_per_component - 1) / block_dim_per_component,
                     block_dim_per_component>>>(d_corner_weights, d_planar_weights, particles_size,
                                                sigma_corner_matching, sigma_planar_matching);
  hipDeviceSynchronize();

  err = hipMemcpy(h_corner_weights, d_corner_weights, particles_size * sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess)
  {
    fprintf(stderr,
            "Failed to copy vector output corners from device to host (localization and mapping interface) (error code "
            "%s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(h_planar_weights, d_planar_weights, particles_size * sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess)
  {
    fprintf(stderr,
            "Failed to copy vector output planars from device to host (localization and mapping interface) (error code "
            "%s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Save particles' weights
  for (int i = 0; i < particles_size; i++)
  {
    particles[i].w_ = h_corner_weights[i] * h_planar_weights[i];
  }

  // Free allocated memory
  for (int i = 0; i < localization_mapping_interface->number_of_nodes_; i++)
  {
    if (localization_mapping_interface->sizes_[i] > 0)
    {
      hipFree(tmp[i]);
    }
  }
  hipFree(d_array);
  hipFree(d_sizes);
  hipFree(d_info);
  hipFree(d_transformations);
  hipFree(d_corners);
  hipFree(d_planars);
  hipFree(d_corner_weights);
  hipFree(d_planar_weights);
  hipFree(d_look_up_table.look_up_table_);
  free(tmp);
  free(h_transformations);
  free(h_corner_weights);
  free(h_planar_weights);
  free(h_corners);
  free(h_planars);
}
}  // namespace vineslam