#include <chrono>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

namespace vineslam
{
__global__ void generate_random_numbers(int* numbers, int size, int max_n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < size)
  {
    hiprandState state;
    hiprand_init(clock64(), i, 0, &state);
    numbers[i] = ceilf(hiprand_uniform(&state) * max_n);
  }
}

struct sum_functor
{
  int R;
  int C;
  int* arr;

  sum_functor(int _R, int _C, int* _arr) : R(_R), C(_C), arr(_arr){};

  __host__ __device__ int operator()(int myC)
  {
    int sum = 0;
    for (int i = 0; i < R; i++)
      sum += arr[i * C + myC];
    return sum;
  }
};

// Global function means it will be executed on the device (GPU)
__global__ void loop(float* xx, float* yy, float* zz, int* rand_numbers, int* inliers_map, int size, int n_iters,
                     float dist_threshold)
{
  int iteration_idx = threadIdx.x + blockIdx.x * blockDim.x;
  int point_idx = threadIdx.y + blockIdx.y * blockDim.y;

  if (iteration_idx < n_iters)
  {
    if (point_idx < size)
    {
      // Get random indexes
      int r1 = rand_numbers[iteration_idx * 3];
      int r2 = rand_numbers[iteration_idx * 3 + 1];
      int r3 = rand_numbers[iteration_idx * 3 + 2];

      /*
      // Check if we have an inlier
      atomicAdd(
          &inliers_map[iteration_idx],
          (fabs(((yy[r2] - yy[r1]) * (zz[r3] - zz[r1]) - (zz[r2] - zz[r1]) * (yy[r3] - yy[r1])) * xx[point_idx] +
                (-((xx[r2] - xx[r1]) * (zz[r3] - zz[r1]) - (zz[r2] - zz[r1]) * (xx[r3] - xx[r1]))) * yy[point_idx] +
                ((xx[r2] - xx[r1]) * (yy[r3] - yy[r1]) - (yy[r2] - yy[r1]) * (xx[r3] - xx[r1])) * zz[point_idx] +
                (-(((yy[r2] - yy[r1]) * (zz[r3] - zz[r1]) - (zz[r2] - zz[r1]) * (yy[r3] - yy[r1])) * xx[r1] +
                   (-((xx[r2] - xx[r1]) * (zz[r3] - zz[r1]) - (zz[r2] - zz[r1]) * (xx[r3] - xx[r1]))) * yy[r1] +
                   ((xx[r2] - xx[r1]) * (yy[r3] - yy[r1]) - (yy[r2] - yy[r1]) * (xx[r3] - xx[r1])) * zz[r1])))) /
                  sqrt(((yy[r2] - yy[r1]) * (zz[r3] - zz[r1]) - (zz[r2] - zz[r1]) * (yy[r3] - yy[r1])) *
                           ((yy[r2] - yy[r1]) * (zz[r3] - zz[r1]) - (zz[r2] - zz[r1]) * (yy[r3] - yy[r1])) +
                       ((xx[r2] - xx[r1]) * (zz[r3] - zz[r1]) - (zz[r2] - zz[r1]) * (xx[r3] - xx[r1])) *
                           ((xx[r2] - xx[r1]) * (zz[r3] - zz[r1]) - (zz[r2] - zz[r1]) * (xx[r3] - xx[r1])) +
                       ((xx[r2] - xx[r1]) * (yy[r3] - yy[r1]) - (yy[r2] - yy[r1]) * (xx[r3] - xx[r1])) *
                           ((xx[r2] - xx[r1]) * (yy[r3] - yy[r1]) - (yy[r2] - yy[r1]) * (xx[r3] - xx[r1]))) <
              dist_threshold);
      */

      // The above code abbreviates the following one
      float x21 = xx[r2] - xx[r1];
      float y21 = yy[r2] - yy[r1];
      float z21 = zz[r2] - zz[r1];
      float x31 = xx[r3] - xx[r1];
      float y31 = yy[r3] - yy[r1];
      float z31 = zz[r3] - zz[r1];
      float l_a = +(y21 * z31 - z21 * y31);
      float l_b = -(x21 * z31 - z21 * x31);
      float l_c = +(x21 * y31 - y21 * x31);
      float l_d = -(l_a * xx[r1] + l_b * yy[r1] + l_c * zz[r1]);

      // Check if we have an inlier
      float norm = sqrt(l_a * l_a + l_b * l_b + l_c * l_c);
      float pt_xx = xx[point_idx];
      float pt_yy = yy[point_idx];
      float pt_zz = zz[point_idx];

      atomicAdd(&inliers_map[iteration_idx],
                (fabs(l_a * pt_xx + l_b * pt_yy + l_c * pt_zz + l_d) / norm) < dist_threshold);
    }
  }
}

int singleIteration(float* xx, float* yy, float* zz, float* out_xx, float* out_yy, float* out_zz, int* rand_numbers,
                     int pos, int size, float dist_threshold)
{
  // Get points
  float xx1 = xx[rand_numbers[pos * 3 + 0]];
  float yy1 = yy[rand_numbers[pos * 3 + 0]];
  float zz1 = zz[rand_numbers[pos * 3 + 0]];
  float xx2 = xx[rand_numbers[pos * 3 + 1]];
  float yy2 = yy[rand_numbers[pos * 3 + 1]];
  float zz2 = zz[rand_numbers[pos * 3 + 1]];
  float xx3 = xx[rand_numbers[pos * 3 + 2]];
  float yy3 = yy[rand_numbers[pos * 3 + 2]];
  float zz3 = zz[rand_numbers[pos * 3 + 2]];

  // Extract the plane hessian coefficients
  float x21 = xx2 - xx1;
  float y21 = yy2 - yy1;
  float z21 = zz2 - zz1;
  float x31 = xx3 - xx1;
  float y31 = yy3 - yy1;
  float z31 = zz3 - zz1;
  float l_a = +(y21 * z31 - z21 * y31);
  float l_b = -(x21 * z31 - z21 * x31);
  float l_c = +(x21 * y31 - y21 * x31);
  float l_d = -(l_a * xx1 + l_b * yy1 + l_c * zz1);
  float norm = sqrt(l_a * l_a + l_b * l_b + l_c * l_c);

  // Check if we have an inlier
  int j = 0;
  for (int i = 0; i < size; i++)
  {
    float pt_xx = xx[i];
    float pt_yy = yy[i];
    float pt_zz = zz[i];
    if (fabs(l_a * pt_xx + l_b * pt_yy + l_c * pt_zz + l_d) / norm < dist_threshold)
    {
      out_xx[j] = xx[i];
      out_yy[j] = yy[i];
      out_zz[j] = zz[i];
      j++;
    }
  }

  return j;
}

void ransac(float* xx, float* yy, float* zz, float* out_xx, float* out_yy, float* out_zz, int& size, int n_iters,
            float dist_threshold)
{
  if (size == 0)
  {
    return;
  }

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  int n_threads = n_iters;
  int ssize = size * sizeof(float);

  // Allocate memory for the device vector of points
  float *d_xx = NULL, *d_yy = NULL, *d_zz = NULL;

  err = hipMalloc((void**)&d_xx, ssize);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector xx (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMalloc((void**)&d_yy, ssize);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector yy (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMalloc((void**)&d_zz, ssize);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device vector zz (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Inliers array
  int *d_inliers_map, *inliers_map;
  inliers_map = (int*)malloc(n_iters * sizeof(int));
  for (int i = 0; i < n_iters; i++)
  {
    inliers_map[i] = 0;
  }
  err = hipMalloc((void**)&d_inliers_map, n_iters * sizeof(int));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device inliers_map vector (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Random numbers array
  int *d_rand_numbers, *rand_numbers;
  int n_rands = n_iters * 3;
  rand_numbers = (int*)malloc(n_rands * sizeof(int));
  err = hipMalloc((void**)&d_rand_numbers, n_rands * sizeof(int));
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to allocate device rand_numbers vector (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy all the data to the device
  err = hipMemcpy(d_xx, xx, ssize, hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector xx from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_yy, yy, ssize, hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector yy from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_zz, zz, ssize, hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector zz from host to device (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_inliers_map, inliers_map, n_iters * sizeof(int), hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector inliers_map from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Generate all the random numbers required for ransac
  generate_random_numbers<<<(n_rands + n_threads - 1) / n_threads, n_threads>>>(d_rand_numbers, n_rands, size);
  err = hipMemcpy(rand_numbers, d_rand_numbers, n_rands * sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector rand_numbers from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Call ransac
  int block_dim_per_component = 8;
  dim3 block_dim(block_dim_per_component, block_dim_per_component);
  dim3 grid_dim((n_iters + block_dim.x - 1) / block_dim.x, (size + block_dim.y - 1) / block_dim.y);
  loop<<<grid_dim, block_dim>>>(d_xx, d_yy, d_zz, d_rand_numbers, d_inliers_map, size, n_iters, dist_threshold);
  hipDeviceSynchronize();

  // Get iteration with most inliers
  err = hipMemcpy(inliers_map, d_inliers_map, n_iters * sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Failed to copy vector inliers_map from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  int max = 0;
  int pos = -1;
  for (int i = 0; i < n_iters; i++)
  {
    if (inliers_map[i] > max)
    {
      max = inliers_map[i];
      pos = i;
    }
  }

  // Compute the inliers given the iteration number
  if (pos > 0)
  {
    int res = singleIteration(xx, yy, zz, out_xx, out_yy, out_zz, rand_numbers, pos, size, dist_threshold);
    if (res != max)
    {
      size = 0;
#if VERBOSE == 1
      std::cout << "ERROR (Ransac GPU) - singleIteration does not match kernel result (" << max << "," << res << ") ... \n" << std::flush;
#endif
    }
    else
    {
      size = max;
    }
  }
  else
  {
    size = 0;
  }

  hipFree(d_rand_numbers);
  hipFree(d_inliers_map);
  hipFree(d_xx);
  hipFree(d_yy);
  hipFree(d_zz);
  free(rand_numbers);
  free(inliers_map);
}
}  // namespace vineslam